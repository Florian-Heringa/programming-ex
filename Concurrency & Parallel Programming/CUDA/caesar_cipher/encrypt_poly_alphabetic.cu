#include "hip/hip_runtime.h"
/* Niek Kabel (11031174) and Florian Heringa. */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.
   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut, int *keys, int num_keys) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceDataOut[index] = deviceDataIn[index] + keys[index % num_keys];
}

__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut, int *keys, int num_keys) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceDataOut[index] = deviceDataIn[index] - keys[index % num_keys];
}

int fileSize() {
  int size; 

  ifstream file ("original.data", ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.close();
  }
  else {
    cout << "Unable to open file";
    size = -1; 
  }
  return size; 
}

int readData(char *fileName, char *data) {

  streampos size;

  ifstream file (fileName, ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.seekg (0, ios::beg);
    file.read (data, size);
    file.close();

    cout << "The entire file content is in memory." << endl;
  }
  else cout << "Unable to open file" << endl;
  return 0;
}

int writeData(int size, char *fileName, char *data) {
  ofstream file (fileName, ios::out|ios::binary|ios::trunc);
  if (file.is_open())
  {
    file.write (data, size);
    file.close();

    cout << "The entire file content was written to file." << endl;
    return 0;
  }
  else cout << "Unable to open file";

  return -1; 
}

int EncryptSeq (int n, char* data_in, char* data_out, int *keys, int num_keys) 
{  
  int i;
  timer sequentialTime = timer("Sequential encryption");
  
  sequentialTime.start();
  for (i=0; i<n; i++) { data_out[i]=data_in[i] + keys[i % num_keys]; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;
  
  return 0; 
}

int DecryptSeq (int n, char* data_in, char* data_out, int *keys, int num_keys)
{
  int i;
  timer sequentialTime = timer("Sequential decryption");

  sequentialTime.start();
  for (i=0; i<n; i++) { data_out[i]=data_in[i] - keys[i % num_keys]; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

  return 0;
}


int EncryptCuda (int n, char* data_in, char* data_out, int *keys, int num_keys) {
    int threadBlockSize = 512;

    // Allocate blocks for the plaintext, ciphertext and keys on the GPU.
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    int* deviceKeys = NULL;
    checkCudaCall(hipMalloc((void **) &deviceKeys, num_keys * sizeof(int)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        checkCudaCall(hipFree(deviceDataOut));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // Copy the plaintext and keys to the GPU.
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceKeys, keys, num_keys*sizeof(int), hipMemcpyHostToDevice));
    memoryTime.stop();

    // Each kernel will shift a character of the ciphertext right by key positions.
    // For each next character, the keys are rotated.
    // The remainder of characters not shifted right by the GPU are shifted right
    // by the CPU, which is always less than one threadblock.
    kernelTime1.start();
    encryptKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceDataIn, deviceDataOut, deviceKeys, num_keys);
    for (int i = n/threadBlockSize * threadBlockSize; i < n; i++) {
      data_out[i] = data_in[i] + keys[i % num_keys];
    }
    hipDeviceSynchronize();
    kernelTime1.stop();

    // Check whether the kernel invocation was successful.
    checkCudaCall(hipGetLastError());

    // Copy the ciphertext back.
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n/threadBlockSize * threadBlockSize * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));
    checkCudaCall(hipFree(deviceKeys));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int DecryptCuda (int n, char* data_in, char* data_out, int *keys, int num_keys) {
    int threadBlockSize = 512;

    // Allocate blocks for the ciphertext, plaintext and keys on the GPU.
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    int* deviceKeys = NULL;
    checkCudaCall(hipMalloc((void **) &deviceKeys, num_keys * sizeof(int)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        checkCudaCall(hipFree(deviceDataOut));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // Copy the ciphertext and keys to the GPU.
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceKeys, keys, num_keys*sizeof(int), hipMemcpyHostToDevice));
    memoryTime.stop();

    // Each kernel will shift a character of the ciphertext left by key positions.
    // For each next character, the keys are rotated.
    // The remainder of characters not shifted left by the GPU are shifted left
    // by the CPU, which is always less than one threadblock.
    kernelTime1.start();
    decryptKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceDataIn, deviceDataOut, deviceKeys, num_keys);
    for (int i = n/threadBlockSize * threadBlockSize; i < n; i++) {
      data_out[i] = data_in[i] - keys[i % num_keys];
    }
    hipDeviceSynchronize();
    kernelTime1.stop();

    // Check whether the kernel invocation was successful.
    checkCudaCall(hipGetLastError());

    // Copy the plaintext back.
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n/threadBlockSize * threadBlockSize * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));
    checkCudaCall(hipFree(deviceKeys));

    cout << fixed << setprecision(6);
    cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int main(int argc, char* argv[]) {
    int n, num_keys;
    int *keys;

    num_keys = argc - 1;
    keys = new int[num_keys];

    for (int i = 1; i < argc; i++) {
      keys[i] = atoi(argv[i]);
    }

    if (argc < 2) {
      cout << "Usage: " << argv[0] << " key1 key2 .." << endl;
      exit(0);
    }

    n = fileSize();
    if (n == -1) {
      cout << "File not found! Exiting ... " << endl; 
      exit(0);
    }

    char* data_in = new char[n];
    char* data_out = new char[n];    
    readData("original.data", data_in);

    // When the same letter in the alphabet is substituted by
    // the same letter in the cipher alphabet, the frequency is preserved.
    // Given that each letter has a different relative frequency,
    // by frequency analysis, we can break monoalphabetic codes when
    // we know its language and only for large ciphertexts (accurate frequency).
    // The ongoing battle between code makers and breakers produced
    // a polyalphabetic cipher where the cipher alhabet changes.
    // We implemented a polyalphabetic substitution cipher,
    // consisting of an arbitrary number of shifted cipher alphabets
    // by subsequent command-line provided keys.

    cout << "Encrypting a file of " << n << " characters." << endl;

    EncryptSeq(n, data_in, data_out, keys, num_keys);
    writeData(n, "sequential.data", data_out);
    EncryptCuda(n, data_in, data_out, keys, num_keys);
    writeData(n, "cuda.data", data_out);  

    readData("cuda.data", data_in);

    cout << "Decrypting a file of " << n << " characters" << endl;
    DecryptSeq(n, data_in, data_out, keys, num_keys);
    writeData(n, "sequential_decrypted.data", data_out);
    DecryptCuda(n, data_in, data_out, keys, num_keys);
    writeData(n, "recovered.data", data_out); 
 
    delete[] keys;
    delete[] data_in;
    delete[] data_out;
    
    return 0;
}

