#include "hip/hip_runtime.h"
/* Niek Kabel (11031174) and Florian Heringa. */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.
   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut, int key) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceDataOut[index] = deviceDataIn[index] + key;
}

__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut, int key) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceDataOut[index] = deviceDataIn[index] - key;
}

int fileSize() {
  int size; 

  ifstream file ("original.data", ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.close();
  }
  else {
    cout << "Unable to open file";
    size = -1; 
  }
  return size; 
}

int readData(char *fileName, char *data) {

  streampos size;

  ifstream file (fileName, ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.seekg (0, ios::beg);
    file.read (data, size);
    file.close();

    cout << "The entire file content is in memory." << endl;
  }
  else cout << "Unable to open file" << endl;
  return 0;
}

int writeData(int size, char *fileName, char *data) {
  ofstream file (fileName, ios::out|ios::binary|ios::trunc);
  if (file.is_open())
  {
    file.write (data, size);
    file.close();

    cout << "The entire file content was written to file." << endl;
    return 0;
  }
  else cout << "Unable to open file";

  return -1; 
}

int EncryptSeq (int n, char* data_in, char* data_out, int key) 
{  
  int i;
  timer sequentialTime = timer("Sequential encryption");
  
  sequentialTime.start();
  for (i=0; i<n; i++) { data_out[i]=data_in[i] + key; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;
  
  return 0; 
}

int DecryptSeq (int n, char* data_in, char* data_out, int key)
{
  int i;
  timer sequentialTime = timer("Sequential decryption");

  sequentialTime.start();
  for (i=0; i<n; i++) { data_out[i]=data_in[i] - key; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

  return 0;
}


int EncryptCuda (int n, char* data_in, char* data_out, int key) {
    int threadBlockSize = 512;

    // Allocate blocks for the plaintext and ciphertext on the GPU.
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // Copy the plaintext to the GPU.
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // Each kernel will shift a character of the ciphertext right by key positions.
    // The remainder of characters not shifted right by the GPU are shifted right
    // by the CPU, which is always less than one threadblock.
    kernelTime1.start();
    encryptKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceDataIn, deviceDataOut, key);
    for (int i = n/threadBlockSize * threadBlockSize; i < n; i++) {
      data_out[i] = data_in[i] + key;
    }
    hipDeviceSynchronize();
    kernelTime1.stop();

    // Check whether the kernel invocation was successful.
    checkCudaCall(hipGetLastError());

    // Copy the ciphertext back.
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n/threadBlockSize * threadBlockSize * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int DecryptCuda (int n, char* data_in, char* data_out, int key) {
    int threadBlockSize = 512;

    // Allocate blocks for the ciphertext and plaintext on the GPU.
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // Copy the ciphertext to the GPU.
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // Each kernel will shift a character of the ciphertext left by key positions.
    // The remainder of characters not shifted left by the GPU are shifted left
    // by the CPU, which is always less than one threadblock.
    kernelTime1.start();
    decryptKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceDataIn, deviceDataOut, key);
    for (int i = n/threadBlockSize * threadBlockSize; i < n; i++) {
      data_out[i] = data_in[i] - key;
    }
    hipDeviceSynchronize();
    kernelTime1.stop();

    // Check whether the kernel invocation was successful.
    checkCudaCall(hipGetLastError());

    // Copy the plaintext back.
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n/threadBlockSize * threadBlockSize * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int main(int argc, char* argv[]) {
    int n, key;

    if (argc < 2) {
      cout << "Usage: " << argv[0] << " key" << endl;
      exit(0);
    }
    key = atoi(argv[1]);

    n = fileSize();
    if (n == -1) {
      cout << "File not found! Exiting ... " << endl;
      exit(0);
    }

    char* data_in = new char[n];
    char* data_out = new char[n];
    readData("original.data", data_in);

    // The art of secret writing consists of stenography (hiding the message)
    // and cryptography (hiding the meaning of a message by transpositioning
    // and/or substituting characters (enciphering) or words (encrypting),
    // described by the cryptographic algorithm and (a)symmetric key, both of
    // which must be known for decrypting unless we enter the realm of code breaking).
    // We will implement a monoalphabetic substitution cipher called Caesar's
    // cipher but not constrained to only a-z to prevent word counting and such.

    /* Encipherment. */
    cout << "Encrypting a file of " << n << " characters." << endl;

    EncryptSeq(n, data_in, data_out, key);
    writeData(n, "sequential.data", data_out);
    EncryptCuda(n, data_in, data_out, key);
    writeData(n, "cuda.data", data_out);  

    readData("cuda.data", data_in);

    /* Decipherment. */
    cout << "Decrypting a file of " << n << "characters" << endl;
    DecryptSeq(n, data_in, data_out, key);
    writeData(n, "sequential_decrypted.data", data_out);
    DecryptCuda(n, data_in, data_out, key); 
    writeData(n, "recovered.data", data_out); 
 
    delete[] data_in;
    delete[] data_out;
    
    return 0;
}
