#include "hip/hip_runtime.h"
/* Niek Kabel (11031174) and Florian Heringa. */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

int fileSize() {
  int size; 

  ifstream file ("original.data", ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.close();
  }
  else {
    cout << "Unable to open file";
    size = -1; 
  }
  return size; 
}

int readData(char *fileName, char *data) {

  streampos size;

  ifstream file (fileName, ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.seekg (0, ios::beg);
    file.read (data, size);
    file.close();

    cout << "The entire file content is in memory." << endl;
  }
  else cout << "Unable to open file" << endl;
  return 0;
}

/* Utility function, use to do error checking.
   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

__global__ void checksumKernel(char* deviceChilds, char *deviceParents, int threads) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < threads) {
        // We write the parent of two neighbour nodes on the position of
        // the pair's index, thereby constructing a right-angled triangle
        // which will prove usefull for adding dont-care parents.
        deviceParents[index] = deviceChilds[2 * index] + deviceChilds[2 * index + 1];
        // We can simply always add a dont-care parent to avoid GPU branching
        // because it is always written in the second half which we disregard
        // in subsequent levels because of the right-angled triangle structure.
        deviceParents[threads] = 0;
    }
}

int checksumCuda (char *filedata, int filesize) {
    int threadBlockSize = 512;

    // Allocate a block for file data on the GPU.
    // We can only neatly replace the childs with their parents if
    // the kernels executed in order. However we do not know the order in
    // which kernels are scheduled to execute and therefore
    // must split the parents from the childs and rotate between them.
    char* deviceChilds = NULL;
    checkCudaCall(hipMalloc((void **) &deviceChilds, filesize * sizeof(char)));
    if (deviceChilds == NULL) {
        cerr << "Could not allocate memory" << endl;
        return -1;
    }
    char* deviceParents = NULL;
    checkCudaCall(hipMalloc((void **) &deviceParents, filesize * sizeof(char)));
    if (deviceParents == NULL) {
        checkCudaCall(hipFree(deviceChilds));
        cerr << "Could not allocate memory" << endl;
        return -1;
    }
    char* tmp;

    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // Copy the file data to the GPU.
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceChilds, filedata, filesize * sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // Execute kernel: we keep reducing from leafs to the root's childs by
    // invoking half as many kernels as nodes, each of which will reduce
    // two neighbour nodes by taking their sum, to construct the parent node.
    // We correct for an odd number of parents by adding a dont-care parent.
    // Initially we assume the number of leafs is divisible by two.
    // Finally we conjecture that the process of dividing a number if even or
    // increment by one if odd will eventually reach one (not as difficult as
    // Collatz conjecture). Its proof proves the finiteness of below loop.
    kernelTime.start();
    for (int threads = filesize / 2; threads > 1; threads /= 2) {
        checksumKernel<<<threads / threadBlockSize + 1, threadBlockSize>>>(deviceChilds, deviceParents, threads);
        if (threads & 1 == 1) {
            threads++;
        }
        hipDeviceSynchronize();
        tmp = deviceChilds;
        deviceChilds = deviceParents;
        deviceParents = tmp;
    }
    kernelTime.stop();

    // Check whether the kernel invocation was successful.
    checkCudaCall(hipGetLastError());

    // Copy the smallest level (the root's childs) of the binary tree back.
    memoryTime.start();
    checkCudaCall(hipMemcpy(filedata, deviceChilds, 2 * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceChilds));
    checkCudaCall(hipFree(deviceParents));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

    return 0;
}

int main(int argc, char* argv[]) {
    int filesize;
    char *filename, *filedata;

    if (argc < 2) {
      cerr << "Usage: " << argv[0] << " filename" << endl;
      exit(1);
    }
    filename = argv[1];

    filesize = fileSize();
    if (!filesize > 0) {
      cerr << "Filename " << filename << " not found or empty" << endl; 
      exit(1);
    }
    // Rule: when bottom-up constructing a binary tree or reducing from
    // leafs (finest-grained) - character values - to root - their total sum,
    // the amount of nodes must be divisible by two.
    // We can only assume this rule always holds when either we
    // 1. correct at every level by adding a dont-care node if necessary (odd number of nodes) or
    // 2. round the number of leafs to the nearest factor of two.
    // The former will introduce more GPU branching while
    // the latter will stress the CPU to GPU already-botleneck communication.
    // We opted for the former because we optimalised the branching away.
    if (filesize & 1 == 1) {
        filesize++;
    }

    filedata = new char[filesize];
    // If we added a dont-care node, make sure its value is irrelevant for the total sum.
    filedata[filesize - 1] = 0;  
    readData(filename, filedata);

    cout << "Calculating the checksum of " << filename << " (" << filesize << " characters)" << endl;

    checksumCuda(filedata, filesize);
    // Reduce the smallest level of the binary tree on the CPU.
    cout << "Checksum: " << (int)(filedata[0] + filedata[1]) << endl;
 
    delete[] filedata;
    
    return 0;
}