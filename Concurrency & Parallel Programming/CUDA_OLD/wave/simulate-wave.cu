#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"
#include "file.h"

using namespace std;

typedef float (*func_t)(float x);

/*
 * Simple gauss with mu=0, sigma^1=1
 */
float gauss(float x)
{
    return exp((-1 * x * x) / 2);
}

/*
 * Fills a given array with samples of a given function. This is used to fill
 * the initial arrays with some starting data, to run the simulation on.
 *
 * The first sample is placed at array index `offset'. `range' samples are
 * taken, so your array should be able to store at least offset+range doubles.
 * The function `f' is sampled `range' times between `sample_start' and
 * `sample_end'.
 */
void fill(float *array, int offset, int range, float sample_start,
        float sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

/* Utility function, use to do error checking.
   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

__global__ void simulateWaveKernel(float* deviceA, float* deviceB, float* deviceResult, int i_max) {
    const float spatial_impact = 0.15;
    // Do not compute the first and last amplitude points.
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (index < i_max - 1) {
        deviceResult[index] = (2 * deviceB[index] - deviceA[index] + spatial_impact *
            (deviceB[index - 1] - (2 * deviceB[index] - deviceB[index + 1])));
    }
}

void simulateWaveCuda(int i_max, int t_max, float* old, float* current, float* next) {
    int threadBlockSize = 512;

    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, i_max * sizeof(float)));
    if (deviceA == NULL) {
        cerr << "could not allocate memory!" << endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, i_max * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cerr << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, i_max * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cerr << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, old, i_max * sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, current, i_max * sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    for (int t = 1; t < t_max; t++) {
        simulateWaveKernel<<<(i_max - 2) / threadBlockSize + 1, threadBlockSize>>>(deviceA, deviceB, deviceResult, i_max);
        float *cdeviceA = deviceA;
        deviceA = deviceB;
        deviceB = deviceResult;
        deviceResult = cdeviceA;
    }
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(current, deviceResult, i_max * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;
}

int main(int argc, char* argv[]) {
    float *old, *current, *next;
    int i_max, t_max;

    if (argc < 3) {
        cerr << "Usage: " << argv[0] << " i_max t_max [initial_data]" << endl;
        return EXIT_FAILURE;
    }

    i_max = atoi(argv[1]);
    t_max = atoi(argv[2]);

    if (i_max < 3) {
        cerr << "Argument error: i_max should be > 2." << endl;
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        cerr << "Argument error: t_max should be >= 1." << endl;
        return EXIT_FAILURE;
    }

    old = new float[i_max];
    current = new float[i_max];
    next = new float[i_max];

    memset(old, 1, i_max * sizeof(float));
    memset(current, 2, i_max * sizeof(float));
    memset(next, 0, i_max * sizeof(float));

    /* How should we will our first two generations? */
    if (argc > 3) {
        if (strcmp(argv[3], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[3], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[3], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[3], "file") == 0) {
            if (argc < 6) {
                cerr << "No files specified!" << endl;
                return EXIT_FAILURE;
            }
            file_read_float_array(argv[4], old, i_max);
            file_read_float_array(argv[5], current, i_max);
        } else {
            cerr << "Unknown initial mode: " << argv[3] << endl;
            return EXIT_FAILURE;
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    timer simulateWaveTimer("simulate wave timer");
    simulateWaveTimer.start();
    simulateWaveCuda(i_max, t_max, old, current, next);
    simulateWaveTimer.stop();
    cout << simulateWaveTimer;

    file_write_float_array("result.txt", current, i_max);
            
    delete[] old;
    delete[] current;
    delete[] next;
    
    return EXIT_SUCCESS;
}
